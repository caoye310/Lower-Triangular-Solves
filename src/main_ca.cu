#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include "mmio.h"
#include "loadmm.h"
#include "csr_matrix.h"
#include "levels.h"
#include "ca_cuda_solve.cuh"
#include "ilu.h"
#include "ca.h"

// =================== serial solver ===================
void reference_solve_csr(const std::vector<int>& rowptr,
    const std::vector<int>& colidx,
    const std::vector<double>& val,
    const std::vector<double>& b,
    std::vector<double>& x)
{
    int n = static_cast<int>(b.size());               

    for (int i = 0; i < n; ++i)
    {
        double sum = 0.0;
        int    diag_idx = -1;

        for (int p = rowptr[i]; p < rowptr[i + 1]; ++p)
        {
            int j = colidx[p];

            if (j <  i)         sum += val[p] * x[j];   
            else if (j == i)    diag_idx = p;          
        }

        if (diag_idx < 0)
        throw std::runtime_error("missing diagonal at row " + std::to_string(i));

        double a_ii = val[diag_idx];
        if (a_ii == 0.0)
        throw std::runtime_error("zero pivot at row " + std::to_string(i));

        x[i] = (b[i] - sum) / a_ii;
    }
}



// =================== compare results ===================
bool compare_results(const std::vector<double>& ref,
                     const std::vector<double>& gpu,
                     double tol = 1e-6) {
    for (size_t i = 0; i < ref.size(); ++i) {
        //std::cout<<ref[i] - gpu[i]<<std::endl;
        if (std::abs((ref[i] - gpu[i])/ref[i]) > tol) {
            std::cerr << "Mismatch at i=" << i << ": ref=" << ref[i]
                     << ", gpu=" << gpu[i] << "\n";
            return false;
        }
    }
    return true;
}

int main(int argc, char** argv) {
    // ---- Step 1: load matrix
    CSRMatrix A, L, U;
    load_mtx_to_csr(argv[1], A);
    cpu_spilu0(A, L, U);
    int N = L.nrows;
    // ---------- Step‑2 RHS ----------
    std::vector<double> b(N, 1.0);

    // ---- Step 3: serial execution for reference
    std::vector<double> y_ref(L.nrows, 0.0);
    reference_solve_csr(L.rowptr, L.colidx, L.data, b, y_ref);

    // ---- Step 4: Levelset rows to ca_ata
    std::vector<std::vector<int>> levels;
    compute_levels(L, levels);
    // for(int i=0;i<levels[0].size();i++){
    //     std::cout<<levels[0][i]<<" ";
    // }
    // std::cout<<std::endl;
    std::vector<std::vector<std::vector<int>>> ca_levels;
    int s = atoi(argv[2]);
    ca_aggregation(levels, s, ca_levels);

    std::cout << " # DAG levels = " << levels.size() << "\n";
    for (size_t i = 0; i < levels.size(); ++i)
        std::cout << "   level " << i << " has " << levels[i].size() << " rows\n";

    std::cout << " # CA ATA = " << ca_levels.size() << "\n";
    for (size_t i = 0; i < ca_levels.size(); ++i)
        //std::cout << "   ca_level " << i << " has " << ca_levels[i].size() << " rows\n";
            for (size_t j = 0; j < ca_levels[i].size(); ++j)
                std::cout << "   ca_level " << j << " of " << i << " has " << ca_levels[i][j].size() << " rows\n";


    
    // ---- Step 6: launch kernel
    std::vector<double> y(L.nrows, 0.0);

    /* ---------- CUDA DAG Triangular Solve ---------- */
    // Each thread process 1 row
    constexpr int TILE_ROWS = 1; 
    // Max # non-zero value     
    constexpr int TILE_NZ   = 128; 

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    parallel_ca_lower_triangular_solve_cuda<TILE_ROWS, TILE_NZ>(
            L, y, b, ca_levels);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "[CUDA DAG Solve Time] " << ms << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // /* ---------- （可选）用 CPU 做参考结果并比较 ---------- */
    // std::vector<double> y_ref;
    // reference_solve_csr(A.rowptr, A.colidx, A.data, b, y_ref);

    bool ok = compare_results(y_ref, y);
    std::cout << (ok ? "[PASS] GPU == CPU\n" : "[FAIL] mismatch!\n");
    return 0;
}
